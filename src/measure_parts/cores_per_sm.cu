/*
 * Program to show the COMPUTE CAPABILITY of the current device
 *
 * Author: Alvaro Saiz (UC)
 * Version: Jul 2021
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1


int main(int argc, char** argv) 
{
    hipError_t result;
    int device, coresPerSM;
    
    //cudaSetDevice(CURRENT_DEVICE);
    hipGetDevice(&device);
    result = hipDeviceGetAttribute(&coresPerSM, hipDeviceAttributeWarpSize, device);
    if (result != hipSuccess)
        return EXIT_ERROR;
    printf("%d\n", coresPerSM);
    return EXIT_SUCCESSFULLY;
}