/*
 * Program to show the COMPUTE CAPABILITY of the current device
 *
 * Author: Alvaro Saiz (UC)
 * Version: Jul 2021
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1


int main(int argc, char** argv) 
{
    hipError_t resultMajor, resultMinor;
    int device, computeCapabilityMayor, computeCapabilityMinor;
    
    //cudaSetDevice(CURRENT_DEVICE);
    hipGetDevice(&device);
    resultMajor = hipDeviceGetAttribute(&computeCapabilityMayor, hipDeviceAttributeComputeCapabilityMajor, device);
    resultMinor = hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, device);
    if (resultMajor != hipSuccess || resultMinor != hipSuccess)
        return EXIT_ERROR;
    printf("%d.%d\n",computeCapabilityMayor,computeCapabilityMinor);
    return EXIT_SUCCESSFULLY;
}