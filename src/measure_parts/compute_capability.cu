
#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1

int main(int argc, char** argv) 
{
    hipError_t resultMajor, resultMinor;
    int device, computeCapabilityMayor, computeCapabilityMinor;
    
    //cudaSetDevice(CURRENT_DEVICE);
    hipGetDevice(&device);
    resultMajor = hipDeviceGetAttribute(&computeCapabilityMayor, hipDeviceAttributeComputeCapabilityMajor, device);
    resultMinor = hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, device);
    if (resultMajor != hipSuccess || resultMinor != hipSuccess)
        exit(EXIT_ERROR);
    printf("%d.%d\n",computeCapabilityMayor,computeCapabilityMinor);
    exit(EXIT_SUCCESSFULLY);
}  // preguntar