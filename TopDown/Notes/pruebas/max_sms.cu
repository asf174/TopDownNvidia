
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <nppdefs.h>
int main()
{
    int value, value2;
    int device;
    hipGetDevice(&device);
    hipDeviceGetAttribute(&value, hipDeviceAttributeMultiprocessorCount, device);
    hipDeviceGetAttribute(&value2, hipDeviceAttributeMaxBlocksPerMultiprocessor, device);
    
    printf("%d, %d\n", value, value2);
}