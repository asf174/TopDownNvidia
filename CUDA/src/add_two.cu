#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 3

__global__ void add(int a, int b, int* result) {
	*result = a + b;
}

int main() {

	int* operation = (int *) malloc(N*sizeof(int));
	operation[0] = 2;
	operation[1] = 200;

	// direcciones de los operandos del DEVICE
	int *d_a, *d_b, *d_r;

	// reservo memoria en DEVICE
	hipMalloc((void **) &d_a,sizeof(int)); 
	hipMalloc((void **) &d_b,sizeof(int));
	hipMalloc((void **) &d_r,sizeof(int));


	// copio la memoria del HOST en la de DEVICE
	//cudaMemcpy(destino (device), origen (host), tamanho, cudaMemcpyHostToDevice);
	hipMemcpy(d_a, &operation[0], sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &operation[1], sizeof(int), hipMemcpyHostToDevice);

	// llamo funcion device
	add<<<1,1>>>(operation[0],operation[1],d_r);

	hipDeviceSynchronize();
	// paso el resultado de memoria de DEVICE  a memoria de HOST
	int result;
	hipMemcpy(&result, d_r, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("El resultado es %d\n",result);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_r);
	free(operation);

}
