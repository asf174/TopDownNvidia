
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>


__global__ void printHola() {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	printf("Hola mundo desde el device con thread: %d\n",idx);
	// printf para evitar que salga warning al compilar
}

int main() {
	//printf("Hola Mundo desde el host\n");

	hipProfilerStart();
	printHola<<<1,65>>>();
	hipDeviceSynchronize();
	hipProfilerStop();
	return 0;
}
