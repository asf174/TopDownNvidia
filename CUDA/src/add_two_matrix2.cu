#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef N 
	#define N 3500
#endif

#ifndef numThreadsPerBlock
	#define numThreadsPerBlock 256
#endif

#ifndef numBlock
	#define numBlock (ceil( (float) N*N/numThreadsPerBlock))
#endif

__global__ void addMatrix(int* a, int* b, int* result, int size)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	if (idx < size)
		result[idx] = a[idx] + b[idx];
}

// print matrix indicated by argument
void 
printMtx(int * matrix)
{
	int i,j;
	for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
            printf("%d\t", matrix[i*N+j]);
        printf("\n");
    }
}

double time() {
	struct timeval time;

  	/* take time of execution */
  	gettimeofday(&time,NULL);
	return time.tv_sec*1000.0 + time.tv_usec/1000.0;
}

int
main(int argc, char* argv[])
{

	// create events to measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// another way to measure time
	double initime = time();

	int *matrixA,*matrixB,*matrixResult;
	
	// create matrix
	for (int i = 0; i < N; i++) {
		matrixA = (int *) malloc(N * N * sizeof(int));
		matrixB = (int *) malloc(N * N * sizeof(int));
		matrixResult = (int *) malloc(N * N * sizeof(int));
	}
	for(int i = 0; i < N*N; i++) {
			matrixA[i] = 4;
			matrixB[i] = 10;
	}
	

	// allocate memory in device
	int *matrixA_d, *matrixB_d, *matrixResult_d;

	hipMalloc((void **) &matrixA_d, N * N * sizeof(int));
	hipMalloc((void **) &matrixB_d, N * N * sizeof(int));
	hipMalloc((void **) &matrixResult_d, N * N * sizeof(int));

	hipMemcpy(matrixA_d, matrixA, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(matrixB_d, matrixB, N * N * sizeof(int), hipMemcpyHostToDevice);
		 
	hipEventRecord(start);
	
	hipProfilerStart();
	addMatrix<<<numBlock,numThreadsPerBlock>>>(matrixA_d,matrixB_d,matrixResult_d,N*N);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipProfilerStop();
	hipEventRecord(stop);
	double endtime = time();
	
	hipMemcpy(matrixResult,matrixResult_d,N*N*sizeof(int),hipMemcpyDeviceToHost);

	//printMtx(matrixResult);
	//printMtx(matrixA);
	//printMtx(matrixB);
	//printMtx(matrixResult);

	hipFree(matrixA_d);
	hipFree(matrixB_d);
	hipFree(matrixResult_d);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("NUMBLOCKS: %d THREADS_PER_BLOCK: %d\n", (int) numBlock, (int) numThreadsPerBlock);
	//printf("Time elapsed in DEVICE: %f milliseconds / %g seconds\n",milliseconds, milliseconds/1000);
	//printf("Time elapsed in DEVICE (%d,%d) N = %d : %g milliseconds / %g seconds\n", numBlock,numThreadsPerBlock,N,
	//endtime - initime,(endtime - initime)/1000);
	//printf("%g %d\n",endtime - initime,numThreadsPerBlock);
}