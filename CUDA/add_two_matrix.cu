
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#ifndef N 
	#define N 3500
#endif

#ifndef numBlock
	#define numBlock 1
#endif

#ifndef numThreadsPerBlock
	#define numThreadsPerBlock 2
#endif	

__global__ void addMatrix(int* a, int* b, int* result, int block)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	//printf("Hola mundo desde el device con thread: %d\n",idx);
	for (int i = block*idx; i <block*(idx + 1); i++)
			result[i] = a[i] + b[i];
}

// print matrix indicated by argument
void 
printMtx(int * matrix)
{
	int i,j;
	for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
            printf("%d\t", matrix[i*N+j]);
        printf("\n");
    }
}

double time() {
	struct timeval time;

  	/* take time of execution */
  	gettimeofday(&time,NULL);
	return time.tv_sec*1000.0 + time.tv_usec/1000.0;
}

int
main(int argc, char* argv[])
{

	// create events to measure time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// another way to measure time
	double initime = time();

	int *matrixA,*matrixB,*matrixResult;
	
	// create matrix
	
	for (int i = 0; i < N; i++) {
		matrixA = (int *) malloc(N * N* sizeof(int));
		matrixB = (int *) malloc(N * N* sizeof(int));
		matrixResult = (int *) malloc(N * N * sizeof(int));
	}
	for(int i = 0; i < N*N; i++) {
			matrixA[i] = 4;
			matrixB[i] = 10;
	}
	

	// allocate memory in device
	int *matrixA_d, *matrixB_d, *matrixResult_d;

	hipMalloc((void **) &matrixA_d, N * N * sizeof(int));
	hipMalloc((void **) &matrixB_d, N * N * sizeof(int));
	hipMalloc((void **) &matrixResult_d, N * N * sizeof(int));

	hipMemcpy(matrixA_d,matrixA,N * N * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(matrixB_d,matrixB,N * N * sizeof(int),hipMemcpyHostToDevice);
		 
	hipEventRecord(start);
	addMatrix<<<numBlock,numThreadsPerBlock>>>(matrixA_d,matrixB_d,matrixResult_d,N*N/numBlock*numThreadsPerBlock);


	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop);
	double endtime = time();
	
	hipMemcpy(matrixResult,matrixResult_d,N*N*sizeof(int),hipMemcpyDeviceToHost);

	//printMtx(matrixResult);

	hipFree(matrixA_d);
	hipFree(matrixB_d);
	hipFree(matrixResult_d);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	//printf("Time elapsed in DEVICE: %f milliseconds / %g seconds\n",milliseconds, milliseconds/1000);
	printf("Time elapsed in DEVICE (%d,%d) N = %d : %g milliseconds / %g seconds\n", numBlock,numThreadsPerBlock,N,
	endtime - initime,(endtime - initime)/1000);
}

/* SOME TIME measureS
	Time elapsed in DEVICE (1,2) N = 3500 : 962.264 milliseconds / 0.962264 seconds

*/