
#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1

int main(int argc, char** argv)
{
    hipError_t result;
    int device, frequency;

    //cudaSetDevice(CURRENT_DEVICE);
    hipGetDevice(&device);
    result = hipDeviceGetAttribute(&frequency, hipDeviceAttributeClockRate, device);
    
    if (result != hipSuccess)
        exit(EXIT_ERROR);
    printf("%d\n",result);
    exit(EXIT_SUCCESSFULLY);
}  // preguntar



