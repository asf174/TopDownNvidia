
#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1

int main(int argc, char** argv)
{
    hipError_t result;
    int device, freq_khz;
    
    hipGetDevice(&device);
    result = hipDeviceGetAttribute(&freq_khz, hipDeviceAttributeClockRate, device);
    if (result != hipSuccess)
        exit(EXIT_ERROR);
    int freq_hz = (long long int) freq_khz * 1000;  // Convert from KHz.
    printf("%d\n", freq_hz);
    exit(EXIT_SUCCESSFULLY);
}  // preguntar



