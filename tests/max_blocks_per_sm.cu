/*
 * Program to show the COMPUTE CAPABILITY of the current device
 *
 * Author: Alvaro Saiz (UC)
 * Version: July-2021
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
//#define CURRENT_DEVICE 1
#define EXIT_SUCCESSFULLY 0
#define EXIT_ERROR -1


int main(int argc, char** argv) 
{
    hipError_t result;
    int device, blocksPerSM;
    
    //cudaSetDevice(CURRENT_DEVICE);
    hipGetDevice(&device);
    result = hipDeviceGetAttribute(&blocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor, device);
    if (result != hipSuccess)
        return EXIT_ERROR;
    printf("%d\n", blocksPerSM);
    return EXIT_SUCCESSFULLY;
}